#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>
#include <iostream>
#include <ctime>
#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "../config.h"

/* -- Types ----------------------------------------------------------------- */

typedef struct {
    hiprandState* states[8];
} config;

/* -- Prototypes ------------------------------------------------------------ */

void            vanity_setup(config& vanity);
void            vanity_run(config& vanity);
void __global__ vanity_init(unsigned long long int* seed, hiprandState* state);
void __global__ vanity_scan(hiprandState* state, int* gpu, int* exec_count, int* keys_found, int* resultCount, KeyRecord* results);
bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz);

/* -- Pattern Matching Functions -------------------------------------------- */

__device__ bool check_starts_with(const char* address, const char* pattern) {
    while (*pattern) {
        if (*pattern != '?' && *pattern != *address) {
            return false;
        }
        pattern++;
        address++;
    }
    return true;
}

__device__ bool check_ends_with(const char* address, const char* pattern) {
    size_t addr_len = 0;
    size_t pattern_len = 0;
    
    // Get lengths
    while (address[addr_len]) addr_len++;
    while (pattern[pattern_len]) pattern_len++;
    
    if (pattern_len > addr_len) return false;
    
    const char* addr_end = address + addr_len - pattern_len;
    while (*pattern) {
        if (*pattern != '?' && *pattern != *addr_end) {
            return false;
        }
        pattern++;
        addr_end++;
    }
    return true;
}

__device__ bool check_starts_and_ends_with(const char* address, const char* start_pattern, const char* end_pattern) {
    return check_starts_with(address, start_pattern) && check_ends_with(address, end_pattern);
}

__device__ bool check_contains_mnemonic(const char* address) {
    for (int i = 0; i < sizeof(mnemonic_words) / sizeof(mnemonic_words[0]); i++) {
        const char* word = mnemonic_words[i];
        const char* found = address;
        while (*found) {
            bool match = true;
            const char* w = word;
            const char* f = found;
            while (*w) {
                if (*w != *f) {
                    match = false;
                    break;
                }
                w++;
                f++;
            }
            if (match) return true;
            found++;
        }
    }
    return false;
}

__device__ bool check_pattern(const char* address, const pattern_t* pattern) {
    switch (pattern->type) {
        case PATTERN_TYPE_STARTS_WITH:
            return check_starts_with(address, pattern->pattern);
            
        case PATTERN_TYPE_ENDS_WITH:
            return check_ends_with(address, pattern->pattern);
            
        case PATTERN_TYPE_STARTS_AND_ENDS_WITH:
            return check_starts_and_ends_with(address, pattern->pattern, pattern->end_pattern);
            
        case PATTERN_TYPE_MNEMONIC:
            return check_contains_mnemonic(address);
            
        default:
            return false;
    }
}

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
    ed25519_set_verbose(true);

    config vanity;
    vanity_setup(vanity);
    vanity_run(vanity);
}

// SMITH
std::string getTimeStr() {
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

// SMITH - safe? who knows
unsigned long long int makeSeed() {
    unsigned long long int seed = 0;
    char *pseed = (char *)&seed;

    std::random_device rd;

    for(unsigned int b=0; b<sizeof(seed); b++) {
      auto r = rd();
      char *entropy = (char *)&r;
      pseed[b] = entropy[0];
    }

    return seed;
}

/* -- Vanity Setup Function ------------------------------------------------- */
void vanity_setup(config &vanity) {
    printf("GPU: Initializing Memory\n");
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

    for (int i = 0; i < gpuCount; ++i) {
        hipSetDevice(i);
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, i);

        int blockSize = 0, minGridSize = 0, maxActiveBlocksPerSM = 0;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM, vanity_scan, blockSize, 0);

        int totalBlocks = maxActiveBlocksPerSM * device.multiProcessorCount;
        int totalThreads = totalBlocks * blockSize;
        printf("GPU %d: totalThreads = %d\n", i, totalThreads);

        unsigned long long int rseed = makeSeed();
        printf("Initialising from entropy: %llu\n", rseed);

        unsigned long long int* dev_rseed;
        hipMalloc((void**)&dev_rseed, sizeof(unsigned long long int));
        hipMemcpy(dev_rseed, &rseed, sizeof(unsigned long long int), hipMemcpyHostToDevice);

        // Allocate enough hiprandState for all threads.
        hipMalloc((void **)&(vanity.states[i]), totalThreads * sizeof(hiprandState));
        vanity_init<<<maxActiveBlocksPerSM, blockSize>>>(dev_rseed, vanity.states[i]);
        hipFree(dev_rseed);
    }
    printf("END: Initializing Memory\n");
}

/* -- Vanity Run Function --------------------------------------------------- */
void vanity_run(config &vanity) {
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

    // Overall counters
    unsigned long long executions_total = 0;
    unsigned long long keys_found_total = 0;

    // We assume that no more than maxDevices GPUs will be used.
    const int maxDevices = 100;
    // Arrays to hold pointers to per-GPU device counters and result buffers.
    int* dev_executions_this_gpu[maxDevices] = {0};
    int* dev_keys_found[maxDevices] = {0};
    int* dev_resultCount[maxDevices] = {0};
    KeyRecord* dev_results[maxDevices] = {0};

    // Loop over iterations
    for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();

        unsigned long long executions_this_iteration = 0;
        unsigned long long keys_found_this_iteration = 0;

        // Launch kernels on every GPU.
        for (int g = 0; g < gpuCount; ++g) {
            hipSetDevice(g);

            int blockSize = 0, minGridSize = 0, maxActiveBlocksPerSM = 0;
            // Determine occupancy parameters for the vanity_scan kernel.
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM, vanity_scan, blockSize, 0);
            hipDeviceProp_t devProp;
            hipGetDeviceProperties(&devProp, g);
            int totalBlocks = maxActiveBlocksPerSM * devProp.multiProcessorCount;

            // Allocate a device int holding the GPU id.
            int* dev_g = nullptr;
            hipMalloc((void**)&dev_g, sizeof(int));
            hipMemcpy(dev_g, &g, sizeof(int), hipMemcpyHostToDevice);

            // Allocate and initialize the counters.
            hipMalloc((void**)&dev_keys_found[g], sizeof(int));
            hipMalloc((void**)&dev_executions_this_gpu[g], sizeof(int));
            hipMalloc((void**)&dev_resultCount[g], sizeof(int));
            hipMemset(dev_keys_found[g], 0, sizeof(int));
            hipMemset(dev_executions_this_gpu[g], 0, sizeof(int));
            hipMemset(dev_resultCount[g], 0, sizeof(int));

            // Allocate the result buffer for matching keys.
            const int MAX_RESULTS = 10000;
            hipMalloc((void**)&dev_results[g], sizeof(KeyRecord) * MAX_RESULTS);

            // Launch the kernel on GPU g.
            // The kernel has 6 parameters:
            // 1. the hiprandState array,
            // 2. pointer to GPU id,
            // 3. pointer to execution count,
            // 4. pointer to keys-found count,
            // 5. pointer to the resultCount counter,
            // 6. pointer to the KeyRecord results array.
            vanity_scan<<<totalBlocks, blockSize>>>(vanity.states[g],
                                                      dev_g,
                                                      dev_executions_this_gpu[g],
                                                      dev_keys_found[g],
                                                      dev_resultCount[g],
                                                      dev_results[g]);
            hipFree(dev_g);
        }

        hipDeviceSynchronize();
        auto finish = std::chrono::high_resolution_clock::now();

        // Collect the per-GPU counters.
        for (int g = 0; g < gpuCount; ++g) {
            int gpu_keys = 0;
            hipMemcpy(&gpu_keys, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost);
            keys_found_this_iteration += gpu_keys;
            keys_found_total += gpu_keys;

            int gpu_exec = 0;
            hipMemcpy(&gpu_exec, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost);
            executions_this_iteration += (unsigned long long)gpu_exec * ATTEMPTS_PER_EXECUTION;
            executions_total += (unsigned long long)gpu_exec * ATTEMPTS_PER_EXECUTION;

            hipFree(dev_keys_found[g]);
            hipFree(dev_executions_this_gpu[g]);
            hipFree(dev_resultCount[g]);
            hipFree(dev_results[g]);
        }

        std::chrono::duration<double> elapsed = finish - start;
        printf("%s Iteration %d Attempts: %llu in %f sec at %fcps - Total Attempts %llu - keys found %llu\n",
           getTimeStr().c_str(),
           iter + 1,
           executions_this_iteration,
           elapsed.count(),
           executions_this_iteration / elapsed.count(),
           executions_total,
           keys_found_total);

        if (keys_found_total >= STOP_AFTER_KEYS_FOUND) {
            printf("Enough keys found, Done!\n");
            exit(0);
        }
    }

    printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

__global__ void vanity_init(unsigned long long int* rseed, hiprandState* state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(*rseed + id, id, 0, &state[id]);
}

__global__ void vanity_scan(hiprandState* state,
                              int* gpu,             // GPU id pointer
                              int* exec_count,      // Execution counter
                              int* keys_found,      // Keys found counter
                              int* resultCount,     // Atomic counter for results
                              KeyRecord* results)   // Global array for matching keys
{
    // Compute a unique thread id.
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    atomicAdd(exec_count, 1);

    // Compute the number of patterns and their lengths.
    int numPatterns = sizeof(patterns) / sizeof(pattern_t);
    int prefix_letter_counts[MAX_PATTERNS];
    for (int p = 0; p < numPatterns; p++) {
        int count = 0;
        while (patterns[p].pattern[count] != '\0') {
            count++;
        }
        prefix_letter_counts[p] = count;
    }

    // Local ED25519 state and random state.
    ge_p3 A;
    hiprandState localState = state[id];
    unsigned char seed[32] = {0};
    unsigned char publick[32] = {0};
    unsigned char privatek[64] = {0};
    char key[KEY_STRING_SIZE] = {0};

    // Initialize the seed from hiprand.
    for (int i = 0; i < 32; ++i) {
        float rnd = hiprand_uniform(&localState);
        seed[i] = (uint8_t)(rnd * 255);
    }

    // Main loop: try ATTEMPTS_PER_EXECUTION times.
    for (int attempt = 0; attempt < ATTEMPTS_PER_EXECUTION; ++attempt) {
        // --- Derive the key via SHA512 (inlined) ---
        sha512_context md;
        sha512_init(&md);
        sha512_update(&md, seed, 32);
        sha512_final(&md, privatek);

        // ED25519 key clamping.
        privatek[0]  &= 248;
        privatek[31] &= 63;
        privatek[31] |= 64;

        // Compute the public key.
        ge_scalarmult_base(&A, privatek);
        ge_p3_tobytes(publick, &A);

        // Convert public key to a Base58 address.
        size_t keysize = KEY_STRING_SIZE;
        b58enc(key, &keysize, publick, 32);

        // --- Check all patterns ---
        for (int p = 0; p < numPatterns; ++p) {
            bool match = true;
            for (int j = 0; j < prefix_letter_counts[p]; ++j) {
                if ((patterns[p].pattern[j] != '?') && (patterns[p].pattern[j] != key[j])) {
                    match = false;
                    break;
                }
            }
            if (match) {
                // A match was found.
                atomicAdd(keys_found, 1);

                int index = atomicAdd(resultCount, 1);
                for (int j = 0; j < KEY_STRING_SIZE; j++) {
                    results[index].key[j] = key[j];
                }
                for (int j = 0; j < SEED_SIZE; j++) {
                    results[index].seed[j] = seed[j];
                }

                // Build the full 64-byte private key (seed || public key)
                unsigned char fullPrivate[64];
                for (int n = 0; n < SEED_SIZE; n++) {
                    fullPrivate[n] = seed[n];
                }
                for (int n = 0; n < 32; n++) {
                    fullPrivate[n + SEED_SIZE] = publick[n];
                }
                // Print the JSON object with no extraneous whitespace.
                printf("{\"address\":\"%s\",\"private_key\":[", key);
                for (int n = 0; n < 64; n++) {
                    printf("%d", fullPrivate[n]);
                    if (n < 63)
                        printf(",");
                }
                printf("]}\n");
                break; // Stop checking patterns for this attempt.
            }
        }

        // Increment seed using simple counter logic.
        for (int i = 0; i < 32; ++i) {
            if (seed[i] == 255)
                seed[i] = 0;
            else {
                seed[i] += 1;
                break;
            }
        }
    }

    // Save the updated hiprand state.
    state[id] = localState;
}

bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz) {
    const char b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

    const uint8_t* bin = data;
    int carry;
    size_t i, j, high, zcount = 0;
    size_t size;
    
    while (zcount < binsz && !bin[zcount])
        ++zcount;
    
    size = (binsz - zcount) * 138 / 100 + 1;
    uint8_t buf[256];
    memset(buf, 0, size);
    
    for (i = zcount, high = size - 1; i < binsz; ++i, high = j) {
        for (carry = bin[i], j = size - 1; (j > high) || carry; --j) {
            carry += 256 * buf[j];
            buf[j] = carry % 58;
            carry /= 58;
            if (!j) {
                break;
            }
        }
    }
    
    for (j = 0; j < size && !buf[j]; ++j);
    
    if (*b58sz <= zcount + size - j) {
        *b58sz = zcount + size - j + 1;
        return false;
    }
    
    if (zcount) memset(b58, '1', zcount);
    for (i = zcount; j < size; ++i, ++j) b58[i] = b58digits_ordered[buf[j]];
    b58[i] = '\0';
    *b58sz = i + 1;
    
    return true;
}