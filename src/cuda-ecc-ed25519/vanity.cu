#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>
#include <iostream>
#include <ctime>
#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "../config.h"

/* -- Types ----------------------------------------------------------------- */

typedef struct {
    hiprandState* states[8];
} config;

/* -- Prototypes ------------------------------------------------------------ */

void            vanity_setup(config& vanity);
void            vanity_run(config& vanity);
void __global__ vanity_init(unsigned long long int* seed, hiprandState* state);
void __global__ vanity_scan(hiprandState* state, int* gpu, int* exec_count, int* keys_found, int* resultCount, KeyRecord* results);
bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz);

/* -- Pattern Matching Functions -------------------------------------------- */

__device__ bool check_starts_with(const char* address, const char* pattern) {
    while (*pattern) {
        if (*pattern != '?' && *pattern != *address) {
            return false;
        }
        pattern++;
        address++;
    }
    return true;
}

__device__ bool check_ends_with(const char* address, const char* pattern) {
    size_t addr_len = 0;
    size_t pattern_len = 0;
    
    // Get lengths
    while (address[addr_len]) addr_len++;
    while (pattern[pattern_len]) pattern_len++;
    
    if (pattern_len > addr_len) return false;
    
    const char* addr_end = address + addr_len - pattern_len;
    while (*pattern) {
        if (*pattern != '?' && *pattern != *addr_end) {
            return false;
        }
        pattern++;
        addr_end++;
    }
    return true;
}

__device__ bool check_starts_and_ends_with(const char* address, const char* start_pattern, const char* end_pattern) {
    return check_starts_with(address, start_pattern) && check_ends_with(address, end_pattern);
}

__device__ bool check_contains_mnemonic(const char* address) {
    for (int i = 0; i < sizeof(mnemonic_words) / sizeof(mnemonic_words[0]); i++) {
        const char* word = mnemonic_words[i];
        const char* found = address;
        while (*found) {
            bool match = true;
            const char* w = word;
            const char* f = found;
            while (*w) {
                if (*w != *f) {
                    match = false;
                    break;
                }
                w++;
                f++;
            }
            if (match) return true;
            found++;
        }
    }
    return false;
}

__device__ bool check_pattern(const char* address, const pattern_t* pattern) {
    switch (pattern->type) {
        case PATTERN_TYPE_STARTS_WITH:
            return check_starts_with(address, pattern->pattern);
            
        case PATTERN_TYPE_ENDS_WITH:
            return check_ends_with(address, pattern->pattern);
            
        case PATTERN_TYPE_STARTS_AND_ENDS_WITH:
            return check_starts_and_ends_with(address, pattern->pattern, pattern->end_pattern);
            
        case PATTERN_TYPE_MNEMONIC:
            return check_contains_mnemonic(address);
            
        default:
            return false;
    }
}

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
    ed25519_set_verbose(true);

    config vanity;
    vanity_setup(vanity);
    vanity_run(vanity);
}

// SMITH
std::string getTimeStr() {
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

// SMITH - safe? who knows
unsigned long long int makeSeed() {
    unsigned long long int seed = 0;
    char *pseed = (char *)&seed;

    std::random_device rd;

    for(unsigned int b=0; b<sizeof(seed); b++) {
      auto r = rd();
      char *entropy = (char *)&r;
      pseed[b] = entropy[0];
    }

    return seed;
}

/* -- Vanity Setup Function ------------------------------------------------- */
void vanity_setup(config &vanity) {
    printf("GPU: Initializing Memory\n");
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

    for (int i = 0; i < gpuCount; ++i) {
        hipSetDevice(i);
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, i);

        int blockSize = 0, minGridSize = 0, maxActiveBlocksPerSM = 0;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM, vanity_scan, blockSize, 0);

        int totalBlocks = maxActiveBlocksPerSM * device.multiProcessorCount;
        int totalThreads = totalBlocks * blockSize;
        printf("GPU %d: totalThreads = %d\n", i, totalThreads);

        unsigned long long int rseed = makeSeed();
        printf("Initialising from entropy: %llu\n", rseed);

        unsigned long long int* dev_rseed;
        hipMalloc((void**)&dev_rseed, sizeof(unsigned long long int));
        hipMemcpy(dev_rseed, &rseed, sizeof(unsigned long long int), hipMemcpyHostToDevice);

        // Allocate enough hiprandState for all threads.
        hipMalloc((void **)&(vanity.states[i]), totalThreads * sizeof(hiprandState));
        vanity_init<<<maxActiveBlocksPerSM, blockSize>>>(dev_rseed, vanity.states[i]);
        hipFree(dev_rseed);
    }
    printf("END: Initializing Memory\n");
}

void vanity_run(config &vanity) {
    int gpuCount = 0;
    hipGetDeviceCount(&gpuCount);

    unsigned long long executions_total = 0;
    unsigned long long keys_found_total = 0;

    const int maxDevices = 100;
    // Arrays to hold per-GPU device pointer for counters and results.
    int* dev_executions_this_gpu[maxDevices] = {0};
    int* dev_keys_found[maxDevices] = {0};
    int* dev_resultCount[maxDevices] = {0};
    KeyRecord* dev_results[maxDevices] = {0};

    // For each iteration:
    for (int iter = 0; iter < MAX_ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();
        unsigned long long executions_this_iteration = 0;
        unsigned long long keys_found_this_iteration = 0;

        // Launch a kernel on each GPU.
        for (int g = 0; g < gpuCount; ++g) {
            hipSetDevice(g);

            int blockSize = 0, minGridSize = 0, maxActiveBlocksPerSM = 0;
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM, vanity_scan, blockSize, 0);
            hipDeviceProp_t devProp;
            hipGetDeviceProperties(&devProp, g);
            int totalBlocks = maxActiveBlocksPerSM * devProp.multiProcessorCount;

            // Allocate a device int for GPU id.
            int* dev_g = nullptr;
            hipMalloc((void**)&dev_g, sizeof(int));
            hipMemcpy(dev_g, &g, sizeof(int), hipMemcpyHostToDevice);

            // Allocate and zero-initialize the per-GPU counters.
            hipMalloc((void**)&dev_keys_found[g], sizeof(int));
            hipMalloc((void**)&dev_executions_this_gpu[g], sizeof(int));
            hipMalloc((void**)&dev_resultCount[g], sizeof(int));
            hipMemset(dev_keys_found[g], 0, sizeof(int));
            hipMemset(dev_executions_this_gpu[g], 0, sizeof(int));
            hipMemset(dev_resultCount[g], 0, sizeof(int));

            // Allocate the results buffer (an array of KeyRecord).
            const int MAX_RESULTS = 10000;
            hipMalloc((void**)&dev_results[g], sizeof(KeyRecord) * MAX_RESULTS);

            // Launch the kernel on GPU g.
            vanity_scan<<<totalBlocks, blockSize>>>(vanity.states[g],
                                                      dev_g,
                                                      dev_executions_this_gpu[g],
                                                      dev_keys_found[g],
                                                      dev_resultCount[g],
                                                      dev_results[g]);
            hipFree(dev_g);
        }

        // Wait for all kernels to complete.
        hipDeviceSynchronize();
        auto finish = std::chrono::high_resolution_clock::now();

        // Copy back the per-GPU counters.
        for (int g = 0; g < gpuCount; ++g) {
            int temp = 0;
            hipMemcpy(&temp, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost);
            keys_found_this_iteration += temp;

            int gpu_exec = 0;
            hipMemcpy(&gpu_exec, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost);
            executions_this_iteration += (unsigned long long)gpu_exec * ATTEMPTS_PER_EXECUTION;

            // Free per-GPU allocations.
            hipFree(dev_keys_found[g]);
            hipFree(dev_executions_this_gpu[g]);
            hipFree(dev_resultCount[g]);
            hipFree(dev_results[g]);
        }

        executions_total += executions_this_iteration;
        keys_found_total += keys_found_this_iteration;

        std::chrono::duration<double> elapsed = finish - start;
        printf("%s Iteration %d Attempts: %llu in %f sec at %fcps - Total Attempts %llu - keys found %llu\n",
               getTimeStr().c_str(),
               iter + 1,
               executions_this_iteration,
               elapsed.count(),
               executions_this_iteration / elapsed.count(),
               executions_total,
               keys_found_total);

        if (keys_found_total >= STOP_AFTER_KEYS_FOUND) {
            printf("Enough keys found, Done!\n");
            exit(0);
        }
    }
    printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

__global__ void vanity_init(unsigned long long int* rseed, hiprandState* state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Instead of using *rseed for every thread, add the thread id:
    hiprand_init(*rseed + id, id, 0, &state[id]);
}

__global__ void vanity_scan(hiprandState* state,
                              int* gpu,             // GPU id pointer
                              int* exec_count,      // Execution counter
                              int* keys_found,      // Keys found counter
                              int* resultCount,     // Atomic counter for results
                              KeyRecord* results)   // Global array for matching keys
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    atomicAdd(exec_count, 1);

    // Determine number of patterns and compute each pattern’s length.
    int numPatterns = sizeof(patterns) / sizeof(pattern_t);
    int prefix_letter_counts[MAX_PATTERNS];
    for (int p = 0; p < numPatterns; p++) {
        int count = 0;
        while (patterns[p].pattern[count] != '\0') {
            count++;
        }
        prefix_letter_counts[p] = count;
    }

    // Local ED25519 state and random variables.
    ge_p3 A;
    hiprandState localState = state[id];
    unsigned char seed[32] = {0};
    unsigned char publick[32] = {0};
    unsigned char privatek[64] = {0};
    char key[KEY_STRING_SIZE] = {0};

    // Initialize the seed from hiprand and mix in the thread id.
    for (int i = 0; i < 32; ++i) {
        float rnd = hiprand_uniform(&localState);
        // Multiply by 255 to scale to 0..255, then XOR in part of the thread id.
        seed[i] = ((uint8_t)(rnd * 255)) ^ ((id >> (i % 8)) & 0xFF);
    }

    // Main loop: try ATTEMPTS_PER_EXECUTION times.
    for (int attempt = 0; attempt < ATTEMPTS_PER_EXECUTION; ++attempt) {
        // Derive key data via SHA512.
        sha512_context md;
        sha512_init(&md);
        sha512_update(&md, seed, 32);
        sha512_final(&md, privatek);

        // ED25519 key clamping.
        privatek[0]  &= 248;
        privatek[31] &= 63;
        privatek[31] |= 64;

        // Compute the public key.
        ge_scalarmult_base(&A, privatek);
        ge_p3_tobytes(publick, &A);

        // Convert the public key to a Base58-encoded address.
        size_t keysize = KEY_STRING_SIZE;
        b58enc(key, &keysize, publick, 32);

        // Check each pattern.
        for (int p = 0; p < numPatterns; ++p) {
            bool match = true;
            for (int j = 0; j < prefix_letter_counts[p]; ++j) {
                if ((patterns[p].pattern[j] != '?') && (patterns[p].pattern[j] != key[j])) {
                    match = false;
                    break;
                }
            }
            if (match) {
                // A match was found.
                atomicAdd(keys_found, 1);
                int index = atomicAdd(resultCount, 1);
                // Copy the Base58 address and the seed into results.
                for (int j = 0; j < KEY_STRING_SIZE; j++) {
                    results[index].key[j] = key[j];
                }
                for (int j = 0; j < SEED_SIZE; j++) {
                    results[index].seed[j] = seed[j];
                }
                // Build the full 64-byte private key: first 32 bytes = seed, next 32 = public key.
                unsigned char fullPrivate[64];
                for (int n = 0; n < SEED_SIZE; n++) {
                    fullPrivate[n] = seed[n];
                }
                for (int n = 0; n < 32; n++) {
                    fullPrivate[n + SEED_SIZE] = publick[n];
                }
                // Print the result in JSON format with no extra whitespace.
                printf("{\"address\":\"%s\",\"private_key\":[", key);
                for (int n = 0; n < 64; n++) {
                    printf("%d", fullPrivate[n]);
                    if (n < 63) {
                        printf(",");
                    }
                }
                printf("]}\n");
                break; // Stop checking further patterns for this attempt.
            }
        }

        // Increment seed using a simple counter (note: this is not secure for production)
        for (int i = 0; i < 32; ++i) {
            if (seed[i] == 255)
                seed[i] = 0;
            else {
                seed[i] += 1;
                break;
            }
        }
    }

    // Write back the updated random state.
    state[id] = localState;
}

bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz) {
    const char b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

    const uint8_t* bin = data;
    int carry;
    size_t i, j, high, zcount = 0;
    size_t size;
    
    while (zcount < binsz && !bin[zcount])
        ++zcount;
    
    size = (binsz - zcount) * 138 / 100 + 1;
    uint8_t buf[256];
    memset(buf, 0, size);
    
    for (i = zcount, high = size - 1; i < binsz; ++i, high = j) {
        for (carry = bin[i], j = size - 1; (j > high) || carry; --j) {
            carry += 256 * buf[j];
            buf[j] = carry % 58;
            carry /= 58;
            if (!j) {
                break;
            }
        }
    }
    
    for (j = 0; j < size && !buf[j]; ++j);
    
    if (*b58sz <= zcount + size - j) {
        *b58sz = zcount + size - j + 1;
        return false;
    }
    
    if (zcount) memset(b58, '1', zcount);
    for (i = zcount; j < size; ++i, ++j) b58[i] = b58digits_ordered[buf[j]];
    b58[i] = '\0';
    *b58sz = i + 1;
    
    return true;
}